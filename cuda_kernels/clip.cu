#include "hip/hip_runtime.h"
/*
Copyright 2017 the arraydiff authors

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "common.cuh"
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void symm_unit_clip_fwd_f32_kernel(
    uint32_t dim,
    const float *clip,
    const float *x,
    float *y)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    /*float c = clip[0];
    float a = c / max(fabs(c), 1.0f);*/
    float a = clip[0];
    float x_i = x[idx];
    y[idx] = x_i * ((x_i > 0.0f) + a * (x_i < 0.0f));
  }
}

extern "C" void arraydiff_cuda_kernel_symm_unit_clip_fwd_f32(
    size_t dim,
    const float *clip,
    const float *x,
    float *y,
    hipStream_t stream)
{
  symm_unit_clip_fwd_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, clip, x, y);
}

__global__ void symm_unit_clip_param_bwd_f32_atomic_naive_kernel(
    uint32_t dim,
    const float *clip,
    const float *x,
    const float *dy,
    float *grad)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    /*float c = clip[0];
    float u = max(fabs(c), 1.0f);
    float du = 1.0f * (c > 1.0f) - 1.0f * (c < -1.0f);
    float x_i = x[idx];
    atomicAdd(&grad[0], (1.0f / u) * (1.0f - du * c / u) * dy[idx] * x_i * (x_i < 0.0f));*/
    float x_i = x[idx];
    atomicAdd(&grad[0], dy[idx] * x_i * (x_i < 0.0f));
  }
}

__global__ void symm_unit_clip_param_bwd_f32_atomic_fast_kernel(
    uint32_t dim,
    const float *clip,
    const float *x,
    const float *dy,
    float *grad)
{
  __shared__ float cache[1024];
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    /*float c = clip[0];
    float u = max(fabs(c), 1.0f);
    float du = 1.0f * (c > 1.0f) - 1.0f * (c < -1.0f);
    float x_i = x[idx];
    cache[threadIdx.x] = (1.0f / u) * (1.0f - du * c / u) * dy[idx] * x_i * (x_i < 0.0f);*/
    float x_i = x[idx];
    cache[threadIdx.x] = dy[idx] * x_i * (x_i < 0.0f);
  } else {
    cache[threadIdx.x] = 0.0f;
  }
  __syncthreads();
  threadblock1024_reduce_sum_f32(cache);
  if (idx < dim) {
    if (threadIdx.x == 0) {
      atomicAdd(&grad[0], cache[0]);
    }
  }
}

extern "C" void arraydiff_cuda_kernel_symm_unit_clip_param_bwd_nondeterministic_f32(
    size_t dim,
    const float *clip,
    const float *x,
    const float *dy,
    float *grad,
    hipStream_t stream)
{
  symm_unit_clip_param_bwd_f32_atomic_fast_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, clip, x, dy, grad);
}

__global__ void symm_unit_clip_input_bwd_f32_kernel(
    uint32_t dim,
    const float *clip,
    const float *x,
    const float *dy,
    float *dx)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    /*float c = clip[0];
    float a = c / max(fabs(c), 1.0f);*/
    float a = clip[0];
    float x_i = x[idx];
    dx[idx] += dy[idx] * ((x_i > 0.0f) + a * (x_i < 0.0f));
  }
}

extern "C" void arraydiff_cuda_kernel_symm_unit_clip_input_bwd_f32(
    size_t dim,
    const float *clip,
    const float *x,
    const float *dy,
    float *dx,
    hipStream_t stream)
{
  symm_unit_clip_input_bwd_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, clip, x, dy, dx);
}
