/*
Copyright 2017 the arraydiff authors

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void cast_u8_to_f32(
    uint32_t dim,
    const uint8_t *x,
    float *y)
{
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dim) {
    y[idx] = (float)(x[idx]);
  }
}

extern "C" void arraydiff_cuda_kernel_cast_u8_to_f32(
    size_t dim,
    const uint8_t *x,
    float *y,
    hipStream_t stream)
{
  cast_u8_to_f32<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, x, y);
}

__global__ void cast_u8x4_to_f32x4(
    uint32_t dim,
    const uint8_t *x,
    float *y)
{
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx + 4 <= dim) {
    uint32_t i = idx >> 2;
    uchar4 vx_i = ((const uchar4 *)x)[i];
    float4 vy_i = { (float)vx_i.x, (float)vx_i.y, (float)vx_i.z, (float)vx_i.w };
    ((float4 *)y)[i] = vy_i;
  } else if (idx < dim) {
    y[idx] = (float)(x[idx]);
  }
}

extern "C" void arraydiff_cuda_kernel_cast_u8x4_to_f32x4(
    size_t dim,
    const uint8_t *x,
    float *y,
    hipStream_t stream)
{
  cast_u8x4_to_f32x4<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, x, y);
}
