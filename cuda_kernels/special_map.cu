
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void rect_fwd_kernel_f32(
    uint32_t dim,
    const float *x,
    float *y)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x_i = x[idx];
    y[idx] = x_i * (x_i > 0.0f);
  }
}

extern "C" void arraydiff_cuda_kernel_rect_fwd_f32(
    size_t dim,
    const float *x,
    float *y,
    hipStream_t stream)
{
  rect_fwd_kernel_f32<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, x, y);
}

__global__ void rect_bwd_kernel_f32(
    uint32_t dim,
    const float *x,
    const float *dy,
    float *dx)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    dx[idx] = dy[idx] * (x[idx] > 0.0f);
  }
}

extern "C" void arraydiff_cuda_kernel_rect_bwd_f32(
    size_t dim,
    const float *x,
    const float *dy,
    float *dx,
    hipStream_t stream)
{
  rect_bwd_kernel_f32<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, x, dy, dx);
}
